#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<math.h>
#include <iostream>

#include "opencv2/opencv.hpp"
#include <opencv2\core\core.hpp>
#include <opencv2\highgui\highgui.hpp>

using namespace cv;
using namespace std;

#define C_PI 3.141592653589793238462643383279502884197169399375

const string FILEPATH = "lena512.bmp";

void __global__ SwirlCu(int width, int height, int stride, uchar *pRawBitmapOrig, uchar *pBitmapCopy, double factor)
{
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	// Test to see if we're testing a valid pixel
	if (i >= height || j >= width) return;

	double cX = (double)width / 2.0f;
	double cY = (double)height / 2.0f;
	double relY = cY - i;
	double relX = j - cX;
	// relX and relY are points in our UV space

	double originalAngle;
	if (relX != 0)
	{
		originalAngle = atan(abs(relY) / abs(relX));
		if (relX > 0 && relY < 0) originalAngle = 2.0f*C_PI - originalAngle;
		else if (relX <= 0 && relY >= 0) originalAngle = C_PI - originalAngle;
		else if (relX <= 0 && relY < 0) originalAngle += C_PI;
	}
	else
	{
		if (relY >= 0) originalAngle = 0.5f * C_PI;
		else originalAngle = 1.5f * C_PI;
	}

	double radius = sqrt(relX*relX + relY * relY);
	
	// Equation that determines how much to rotate image by
	double newAngle = originalAngle + 1 / (factor*radius + (4.0f / C_PI));

	// Transform source UV coordinates back into bitmap coordinates
	int srcX = (int)(floor(radius * cos(newAngle) + 0.5f));
	int srcY = (int)(floor(radius * sin(newAngle) + 0.5f));
	srcX += cX;
	srcY += cY;
	srcY = height - srcY;
	// Clamp the source to legal image pixel
	if (srcX < 0) srcX = 0;
	else if (srcX >= width) srcX = width - 1;
	if (srcY < 0) srcY = 0;
	else if (srcY >= height) srcY = height - 1;
	
	// Set the pixel color
	pRawBitmapOrig[i*stride / 4 + j] = pBitmapCopy[srcY*stride / 4 + srcX];
}

int main()
{
	Mat image(512, 512, CV_8UC3, cv::Scalar::all(0));
	
	image = imread(FILEPATH);
	namedWindow("Display window", WINDOW_AUTOSIZE);

	if (!image.data)                              
	{
		cout << "Could not open or find the image" << std::endl;
		return -1;
	}

	const int rows = image.rows;
	const int columns = image.cols;
	cout << "ROWS: " << rows << endl;
	cout<<" COLUMNS: " << columns;

	uchar* h_image = image.data;
	uchar* d_image = new uchar[rows * columns];
	uchar* d_imageCopy = new uchar[rows * columns];

	//Copy the data to the device
	hipMemcpy(d_image, h_image, sizeof(uchar*), hipMemcpyHostToDevice);
	hipMemcpy(d_imageCopy, d_image, sizeof(uchar*), hipMemcpyDeviceToDevice);

	SwirlCu <<<16, 16>>> (rows, columns, rows * 4, d_image, d_imageCopy, 0.005f);
	hipDeviceSynchronize();

	// Copy the data back to the host
	hipMemcpy(h_image, d_image, sizeof(uchar*), hipMemcpyDeviceToHost);

	image.data = h_image;

	imshow("Display window", image);

	/*delete[] h_image;
	delete[] d_image;
	delete[] d_imageCopy;*/

	waitKey(5000);											// Wait for a keystroke in the window
	return 0;
}
